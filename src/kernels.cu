#include "hip/hip_runtime.h"
#include "kernels.h"
#include <thrust/device_vector.h>
using namespace cv::cuda;

inline __device__ __host__ int iDivUp( int a, int b )  		{ return (a % b != 0) ? (a / b + 1) : (a / b); }

__global__ void image2Matrix_kernel(int width, int height,  PtrStepSz<uchar3> image, float* matrix){

    const int w = blockIdx.x;
    const int h = blockIdx.y;

    float alpha = 127.5;
    float beta = 0.0078125;
    if (w < width && h < height)
    {
        uchar3 v = image(h,w);
        *(matrix + 0*height*width + h*width + w) = (float(v.z)-alpha)*beta;
        *(matrix + 1*height*width + h*width + w) = (float(v.y)-alpha)*beta;
        *(matrix + 2*height*width + h*width + w) = (float(v.x)-alpha)*beta;
    }

}
__global__ void image2Matrix_with_transpose_kernel(int width, int height,  PtrStepSz<uchar3> image, float* matrix){

    const int w = blockIdx.x;
    const int h = blockIdx.y;
    float alpha = 127.5;
    float beta = 0.0078125;
    if (w < width && h < height)
    {
        uchar3 v = image(w,h); //swap w and h to transpose
        *(matrix + 0*height*width + h*width + w) = (float(v.z)-alpha)*beta;
        *(matrix + 1*height*width + h*width + w) = (float(v.y)-alpha)*beta;
        *(matrix + 2*height*width + h*width + w) = (float(v.x)-alpha)*beta;
    }

}
void gpu_image2Matrix(int width, int height,  cuda::GpuMat & image, float* matrix, hipStream_t &stream)
{
     /*
        image : input image in GpuMat format, WHC arrangement and BGR order
        matrix: gpu float array, CHW and RGB order
    */
    dim3 block(width, height); // width * height blocks, 1 thread each
    image2Matrix_kernel<<<block,1,0,stream>>>(width,height,image,matrix);
}

void gpu_image2Matrix_with_transpose(int width, int height,  cuda::GpuMat & image, float* matrix, hipStream_t &stream)
{
    dim3 block(width, height); // width * height blocks, 1 thread each
    image2Matrix_with_transpose_kernel<<<block,1,0,stream>>>(width,height,image,matrix);
}


//__global__ void crop_and_resize_kernel(int x1, int y1, int x2, int y2, int PtrStepSz<uchar3> image, int* temp_buffer)
//{
//    const int x = blockDim.x*blockIdx.x+threadIdx.x;
//    const int y = blockDim.y*blockIdx.y+threadIdx.y;
//    if(x>=(x2-x1)||y>=(y2-y1))
//        return;
//    uchar3 v = image(y,x);
//    temp_buffer[]
//}
//
//__global__ void generate_batch_kernel(int crop_size, int width, int height, int * boxes_data, PtrStepSz<uchar3> image, float * output_batch)
//{
//    const int box_idx = blockIdx.x*blockDim.x+threadIdx.x;
//    if(!boxes_data||!output_batch)
//        return hipErrorInvalidDevicePointer;
//    if(crop_size==0||width==0||height==0||num==0)
//        return hipErrorInvalidValue;
//
//    int offset = box_idx*4*sizeof(int);
//    //the bbox
//    int x1 = int(boxes_data+offset);
//    int y1 = int(boxes_data+offset+1);
//    int x2 = int(boxes_data+offset+2);
//    int y2 = int(boxes_data+offset+3);
//
//    // the width and height of area to crop
//    int w = x2-x1;
//    int h = y2-y1;
//
//    // total pixels of area to crop
//    int total_pixels = w*h;
//
//    // creat a temp buffer to store
//    float *temp_buffer = new float[total_pixels*3];
//    const dim3 blockDim(8,8);
//    const dim3 gridDim(iDivUp(w,blockDim.x),iDivUp(h,blockDim.y));
//    crop_kernel<<<gridDim,blockDim>>>(x1,y2,x2,y2,image, temp_buffer);
//}
//
//void boxes2bactch(int num, int crop_size, int width, int height, float * boxes_data, cuda::GpuMat image, float * output_batch, float * hipStream_t& stream)
//{
//    generate_batch_kernel<<<num,1,0,stream>>>(crop_size,width, height, boxes_data, image, output_batch);
//}


//__global__ void generatebox_kernel(int width, int height, float * scores, float * location, float pthreshold )
//{
//    const int w = blockIdx.x;
//    const int h = blockIdx.y;
//    if(w<width && h< height)
//    {
//        float score  = *(scores + width*height + w*height+ h );
//        if(score > pthreshold)
//        {
//
//        }
//    }
//
//
//}
//void gpu_generatebox(int width , int height , void * score, void * location, float scale, float pthreshold)
//{
//
//    int stride = 2;
//    int cellsize = 12;
//    int count = 0;
//    //score p
//    void *p = (float*)score + width * height;
//    void *plocal = (float*)location;
//    struct Bbox bbox;
//    struct orderScore order;
//    for (int row = 0; row < score->height; row++) {
//        for (int col = 0; col < score->width; col++) {
//            if (*p > Pthreshold) {
//                bbox.score = *p;
//                order.score = *p;
//                order.oriOrder = count;
//                bbox.x1 = round((stride * row + 1) / scale);
//                bbox.y1 = round((stride * col + 1) / scale);
//                bbox.x2 = round((stride * row + 1 + cellsize) / scale);
//                bbox.y2 = round((stride * col + 1 + cellsize) / scale);
//                bbox.exist = true;
//                bbox.area = (bbox.x2 - bbox.x1) * (bbox.y2 - bbox.y1);
//                for (int channel = 0; channel < 4; channel++)
//                    bbox.regreCoord[channel] = *(plocal + channel * location->width * location->height);
//                boundingBox_.push_back(bbox);
//                bboxScore_.push_back(order);
//                count++;
//            }
//            p++;
//            plocal++;
//        }
//    }
//    dim3 block(width, height);
//    generatebox_kernel(width, height, score, location, pthreshold);
//}
